#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <sysinfoapi.h>
// #include "cpu_bitmap.h"

#define DIM 512
#define BPP 3

struct v {
  float x, y, z;
 __device__  v operator+(v r) { return v(x + r.x, y + r.y, z + r.z); }
 __device__  v operator*(float r) { return v(x * r, y * r, z * r); }
 __device__  float operator%(v r) { return x * r.x + y * r.y + z * r.z; }
 __device__  v() {}
 __device__  v operator^(v r) {
    return v(y * r.z - z * r.y, z * r.x - x * r.z, x * r.y - y * r.x);
  }
  __device__ v(float a, float b, float c) {
    x = a;
    y = b;
    z = c;
  }
  __device__ v operator!() { return *this * (1 / sqrt(*this % *this)); }
};

__device__ int G[] = {247570, 280596, 280600, 249748, 18578, 18577, 231184, 16, 16};

__device__ int g_seed=1;
__device__ float R() {
  g_seed = (214013*g_seed+2531011);
      return ((g_seed>>16)&0x7FFF) / (float)66635;
}

//The intersection test for line [o,v].
// Return 2 if a hit was found (and also return distance t and bouncing ray n).
// Return 0 if no hit was found but ray goes upward
// Return 1 if no hit was found but ray goes downward
__device__ int TraceRay(v origin, v destination, float &t, v &normal) {
  t = 1e9;
  int m = 0;
  float p = -origin.z / destination.z;
  if (.01 < p) {
      t = p;
      normal = v(0, 0, 1);
      m = 1;
  }

  for (int k = 19; k--;)
    for (int j = 9; j--;)
      if (G[j] & 1 << k) {
        v p = origin + v(-k, 0, -j - 4);
        float b = p % destination;
        float c = p % p - 1;
        float q = b * b - c;

          //Does the ray hit the sphere ?
        if (q > 0) {
          float s = -b - sqrt(q);
            //It does, compute the distance camera-sphere
          if (s < t && s > .01) {
              t = s;
              normal = !(p + destination * t);
              m = 2;
          }
        }
      }
  return m;
}

__device__ v Sample(v origin, v destination, int r) {
  float t;
  v normal;
  if (r > 4) {
    return v();
  }
  int match = TraceRay(origin, destination, t, normal);
  if (!match) {
      //No sphere found and the ray goes upward: Generate a sky color
      return v(.7, .6, 1) * pow(1 - destination.z, 4);
  }

  //A sphere was maybe hit.
  v intersection = origin + destination * t;
  v light_dir = !(v(9 + R(), 9 + R(), 16) + intersection * -1);
  v half_vec = destination + normal * (normal % destination * -2);

  //Calculated the lambertian factor
  float lamb_f = light_dir % normal;

  //Calculate illumination factor (lambertian coefficient > 0 or in shadow)?
  if (lamb_f < 0 || TraceRay(intersection, light_dir, t, normal)) {
      lamb_f = 0;
  }

  float color = pow(light_dir % half_vec * (lamb_f > 0), 99);

  if (match & 1) {
    //No sphere was hit and the ray was going downward: Generate a floor color
    intersection = intersection * .2;
    return ((int)(ceil(intersection.x) + ceil(intersection.y)) & 1 ?
                  v(3, 1, 1) :
                  v(3, 3, 3)) * (lamb_f * .2 + .1);
  }

  //m == 2 A sphere was hit. Cast an ray bouncing from the sphere surface.
  //Attenuate color by 50% since it is bouncing (* .5)
  return v(color, color, color) + Sample(intersection, half_vec, r+1) * .5;
}


__global__ void GetColor(unsigned char *img) {
  int x = blockIdx.x;
  int y = threadIdx.x;
v cam_dir = !v(-6, -16, 0);
v cam_up = !(v(0, 0, 1) ^ cam_dir) * .002;
v cam_right = !(cam_dir ^ cam_up) * .002;
v eye_offset = (cam_up + cam_right) * -256 + cam_dir;
  v color(13, 13, 13);
      for (int r = 64; r--;) {
        v delta = cam_up * (R() - .5) * 99 + cam_right * (R() - .5) * 99;
        color = Sample(v(17, 16, 8) + delta,
                !(delta * -1 + (cam_up * (R() + x) + cam_right * (y + R()) + eye_offset) * 16), 0) * 3.5 + color;
      }
      img[DIM * y * BPP + x * BPP + 0] = color.x;
      img[DIM * y * BPP + x * BPP + 1] = color.y;
      img[DIM * y * BPP + x * BPP + 2] = color.z;
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main() {
  // printf("Starting\n");
  DWORD start_time = GetTickCount();

  // CPUBitmap bitmap( DIM, DIM );
  char* bitmap = new char[DIM * DIM * BPP];
  unsigned char *dev_bitmap;

  // hipMalloc( (void**)&dev_bitmap, bitmap.image_size() );
  hipMalloc( (void**)&dev_bitmap, (DIM * DIM * BPP) );
  // gpuErrchk( hipPeekAtLastError() );

  GetColor<<<DIM,DIM>>>(dev_bitmap); 
  // gpuErrchk( hipDeviceSynchronize() );

  // hipMemcpy( bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost );
  hipMemcpy( bitmap, dev_bitmap, (DIM * DIM * BPP), hipMemcpyDeviceToHost );
  // gpuErrchk( hipPeekAtLastError() );
 
  DWORD elapsed_ms = GetTickCount() - start_time;
  fprintf(stderr, "Time: %dms\n", elapsed_ms);

  printf("P6 512 512 255 ");
  char* c = bitmap;
  for (int y = DIM; y--;) {
    for (int x = DIM; x--;) {
      c = &bitmap[y * DIM * BPP  + x * BPP];
      printf("%c%c%c", c[0], c[1], c[2]);
      c += BPP;
    }
  }

  delete bitmap;
  return EXIT_SUCCESS;
}
