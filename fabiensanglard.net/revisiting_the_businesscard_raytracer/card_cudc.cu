#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <sysinfoapi.h>

#define DIM 512
#define BPP 3

#define INSTRINSIC 1
#if defined(INSTRINSIC)
#define POW __powf
#define CEIL ceilf
#define RSQRT(x) (rsqrtf(x))
#define SQRT(x) (sqrtf(x))
#define DIVIDE(x,y) __fdividef((x),(y))
#else
#define POW pow
#define CEIL ceil
#define RSQRT(x) (1/sqrt(x))
#define SQRT(x) (sqrt((x)))
#define DIVIDE(x,y) ((x)/(y))
#endif

struct v {
  float x, y, z;
 __device__  v operator+(v r) { return v(x + r.x, y + r.y, z + r.z); }
 __device__  v operator*(float r) { return v(x * r, y * r, z * r); }
 __device__  float operator%(v r) {return x * r.x + y * r.y + z * r.z;}
 __device__  v() {}
 __device__  v operator^(v r) {
    return v(y * r.z - z * r.y, z * r.x - x * r.z, x * r.y - y * r.x);
  }
  __device__ v(float a, float b, float c) {
    x = a;
    y = b;
    z = c;
  }
  __device__ v operator!() { return *this * RSQRT(*this % *this); }
};

__device__ int G[] = {247570, 280596, 280600, 249748, 18578, 18577, 231184, 16, 16};

__shared__ int g_seed;
__device__ float R() {
  g_seed = (214013*g_seed+2531011);
      return DIVIDE((g_seed >> 16)&0x7FFF,66635.0f);
}

//The intersection test for line [o,v].
// Return 2 if a hit was found (and also return distance t and bouncing ray n).
// Return 0 if no hit was found but ray goes upward
// Return 1 if no hit was found but ray goes downward
__device__ int TraceRay(v origin, v destination, float &t, v &normal) {
  t = 1e9;
  int m = 0;
  float p = DIVIDE(-origin.z, destination.z);
  if (.01 < p) {
      t = p;
      normal = v(0, 0, 1);
      m = 1;
  }

  for (int k = 19; k--;)
    for (int j = 9; j--;)
      if (G[j] & 1 << k) {
        v p = origin + v(-k, 0, -j - 4);
        float b = p % destination;
        float c = p % p - 1;
        float q = b * b - c;

          //Does the ray hit the sphere ?
        if (q > 0) {
          float s = -b - SQRT(q);
            //It does, compute the distance camera-sphere
          if (s < t && s > 0.01f) {
              t = s;
              normal = !(p + destination * t);
              m = 2;
          }
        }
      }
  return m;
}

__device__ v Sample(v origin, v destination) {
  float attenuation = 1.0f;
  v pixel_color(0,0,0);
  for (int r=0 ; r < 4 ; r++, attenuation /= 2) {
    float t;
    v normal;

    int match = TraceRay(origin, destination, t, normal);
    if (!match) {
        //No sphere found and the ray goes upward: Generate a sky color
        return  pixel_color + v(0.7f, 0.6f, 1.0f) * POW(1 - destination.z, 4) * attenuation;
    }

    //A sphere was maybe hit.
    v intersection = origin + destination * t;
    v light_dir = !(v(9 + R(), 9 + R(), 16) + intersection * -1);
    v half_vec = destination + normal * (normal % destination * -2);

    //Calculated the lambertian factor
    float lamb_f = light_dir % normal;

    //Calculate illumination factor (lambertian coefficient > 0 or in shadow)?
    if (lamb_f < 0 || TraceRay(intersection, light_dir, t, normal)) {
        lamb_f = 0;
    }


    if (match & 1) {
      //No sphere was hit and the ray was going downward: Generate a floor color
      intersection = intersection * .2f;
      v c = ((int)(CEIL(intersection.x) + CEIL(intersection.y)) & 1 ?
                  v(3, 1, 1) :
                  v(3, 3, 3)) * (lamb_f * .2f + .1f);
      return pixel_color + c * attenuation;
    }

    float color = POW(light_dir % half_vec * (lamb_f > 0), 99);
    pixel_color = pixel_color + v(color, color, color) * attenuation;
    //m == 2 A sphere was hit. Cast an ray bouncing from the sphere surface.
    //Attenuate color by 50% since it is bouncing (* .5)
    origin = intersection;
    destination = half_vec;
  }
  return pixel_color;
}


__global__ void GetColor(unsigned char *img) {
  int x = blockIdx.x;
  int y = threadIdx.x;
v cam_dir = !v(-6, -16, 0);
v cam_up = !(v(0, 0, 1) ^ cam_dir) * .002f;
v cam_right = !(cam_dir ^ cam_up) * .002f;
v eye_offset = (cam_up + cam_right) * -256 + cam_dir;
  v color(13, 13, 13);
      for (int r = 64; r--;) {
        v delta = cam_up * (R() - .5) * 99.0f + cam_right * (R() - 0.5f) * 99.0f;
        color = Sample(v(17, 16, 8) + delta,
                !(delta * -1 + (cam_up * (R() + x) + cam_right * (y + R()) + eye_offset) * 16.0f)) * 3.5f + color;
      }
      img[DIM * y * BPP + x * BPP + 0] = color.x;
      img[DIM * y * BPP + x * BPP + 1] = color.y;
      img[DIM * y * BPP + x * BPP + 2] = color.z;
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main() {
  DWORD start_time = GetTickCount();

  // CPUBitmap bitmap( DIM, DIM );
  char* bitmap = new char[DIM * DIM * BPP];
  unsigned char *dev_bitmap;

  // hipMalloc( (void**)&dev_bitmap, bitmap.image_size() );
  hipMalloc( (void**)&dev_bitmap, (DIM * DIM * BPP) );
  gpuErrchk( hipPeekAtLastError() );

  GetColor<<<DIM,DIM>>>(dev_bitmap); 
  gpuErrchk( hipDeviceSynchronize() );

  // hipMemcpy( bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost );
  hipMemcpy( bitmap, dev_bitmap, (DIM * DIM * BPP), hipMemcpyDeviceToHost );
  gpuErrchk( hipPeekAtLastError() );
 
  DWORD elapsed_ms = GetTickCount() - start_time;
  fprintf(stderr, "Time: %dms\n", elapsed_ms);

  printf("P6 512 512 255 ");
  char* c = bitmap;
  for (int y = DIM; y--;) {
    for (int x = DIM; x--;) {
      c = &bitmap[y * DIM * BPP  + x * BPP];
      printf("%c%c%c", c[0], c[1], c[2]);
      c += BPP;
    }
  }

  delete bitmap;
  return EXIT_SUCCESS;
}
