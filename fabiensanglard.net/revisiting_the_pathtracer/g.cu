#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdint.h>
#include <float.h>
#include <limits.h>
#include <stdlib.h>
#include <windows.h>
#include <sysinfoapi.h>
#include <hiprand/hiprand_kernel.h>

#define w  960
#define h  540
#define BPP 3

struct Vec {
  float x;
  float y;
  float z;

  __device__ Vec(float v = 0) {
    x = y = z = v;
  }

  __device__ Vec(float a, float b, float c = 0) {
    x = a;
    y = b;
    z = c;
  }

  __device__  Vec operator+(const Vec r) const  { return Vec(x + r.x , y + r.y , z + r.z); }
  __device__ Vec operator*(const Vec r) const { return   Vec(x * r.x , y * r.y , z * r.z); }
  __device__  float operator%(const Vec r) const {return     x * r.x + y * r.y + z * r.z;}
  __device__ Vec operator!() { return *this * rsqrtf(*this % *this); }
};

__shared__ hiprandState_t states[w];
__device__ float randomVal2() {
   return hiprand_uniform(&states[threadIdx.x]);
} 

__device__ int g_seed=1;
__device__ float randomVal() {
  g_seed = (214013*g_seed+2531011);
      return ((g_seed>>16)&0x7FFF) / (float)66635;
}


// Rectangle CSG equation. Returns minimum signed distance from
// space carved by
// lowerLeft vertex and opposite rectangle vertex upperRight.
__device__ float BoxTest(const Vec& position, Vec lowerLeft, Vec upperRight) {
  lowerLeft = position + lowerLeft * -1.0f;
  upperRight = upperRight + position * -1.0f;
  return -fminf(
      fminf(fminf(lowerLeft.x, upperRight.x), fminf(lowerLeft.y, upperRight.y)),
      fminf(lowerLeft.z, upperRight.z));
}

#define HIT_NONE 0
#define HIT_LETTER 1
#define HIT_WALL 2
#define HIT_SUN 3

// Sample the world using Signed Distance Fields.
__device__ float QueryDatabase(const Vec& position, int &hitType) {
  float distance = 1e9;//FLT_MAX;
  Vec f = position; // Flattened position (z=0)
  f.z = 0;
  
  
  char letters[15*4+1] =               // 15 two points lines
          "5O5_" "5W9W" "5_9_"         // P (without curve)
          "AOEO" "COC_" "A_E_"         // I
          "IOQ_" "I_QO"                // X
          "UOY_" "Y_]O" "WW[W"         // A
          "aOa_" "aWeW" "a_e_" "cWiO"; // R (without curve)

  for (int i = 0; i < sizeof(letters); i += 4) {
    Vec begin = Vec(letters[i] - 79, letters[i + 1] - 79) * .5;
    Vec e = Vec(letters[i + 2] - 79, letters[i + 3] - 79) * .5 + begin * -1;
    Vec o = f + (begin + e * fminf(
                                  -fminf(__fdividef((begin + f * -1) % e , (e % e)), 0),
                                   1)
                             ) * -1;
    distance = fminf(distance, o % o); // compare squared distance.
  }
  distance = sqrtf(distance); // Get real distance, not square distance.
  

  // Two curves (for P and R in PixaR) with hard-coded locations.
  Vec curves[] = {Vec(-11, 6), Vec(11, 6)};
  for (int i = 2; i--;) {
    Vec o = f + curves[i] * -1;
    distance = fminf(distance, o.x > 0 ? fabsf(sqrtf(o % o) - 2)
                                     : (o.y += o.y > 0 ? -2 : 2, sqrtf(o % o)));
  }
  distance = __powf(
    // distance * distance * distance  * distance * 
    // distance * distance *
    // distance  * distance  
    powf(distance, 8)
    + 
    // position.z * position.z * position.z * position.z *
    // position.z * position.z * 
    // position.z * position.z
    powf(position.z, 8)
    , 0.125f) - 0.5f;

  hitType = HIT_LETTER;
  

  float roomDist ;
  roomDist = fminf( // min(A,B) = Union with Constructive solid geometry
                  //-min carves an empty space
      -fminf(       // Lower room
          BoxTest(position, Vec(-30, -.5, -30), Vec(30, 18, 30)),
          // Upper room
          BoxTest(position, Vec(-25, 17, -25), Vec(25, 20, 25))
      ),
      BoxTest( // Ceiling "planks" spaced 8 units apart.
          Vec(fmodf(fabsf(position.x), 8), position.y, position.z),
          Vec(1.5, 18.5, -25), 
          Vec(6.5, 20, 25)
      )
  );
  if (roomDist < distance)
    distance = roomDist, hitType = HIT_WALL;

  float sun = 19.9 - position.y; // Everything above 19.9 is light source.
  if (sun < distance)
    distance = sun, hitType = HIT_SUN;

  return distance;
}

// Perform signed sphere marching
// Returns hitType 0, 1, 2, or 3 and update hit position/normal
__device__ int RayMarching(const Vec& origin, const Vec& direction, Vec& hitPos, Vec& hitNorm) {
  int hitType = HIT_NONE;
  int noHitCount = 0;

  // Signed distance marching
  float d; // distance from closest object in world.
  for (float total_d = 0; total_d < 100; total_d += d) {
    if ((d = QueryDatabase(hitPos = origin + direction * total_d, hitType)) < .01 || ++noHitCount > 99)
      return hitNorm =
                 !Vec(QueryDatabase(hitPos + Vec(.01, 0), noHitCount) - d,
                      QueryDatabase(hitPos + Vec(0, .01), noHitCount) - d,
                      QueryDatabase(hitPos + Vec(0, 0, .01), noHitCount) - d),
             hitType; // Weird return statement where a variable is also
                      // updated.
  }
  return 0;
}

__device__ Vec Trace(Vec origin, Vec direction) {
  Vec sampledPosition;
  Vec normal;
  Vec color = 0;
  Vec attenuation = 1;
  Vec lightDirection(!Vec(0.6f, 0.6f, 1.0f)); // Directional light

  for (int bounceCount = 3; bounceCount--;) {
    int hitType = RayMarching(origin, direction, sampledPosition, normal);
    if (hitType == HIT_NONE)
      break;                     // No hit. This is over, return color.
    if (hitType == HIT_LETTER) { // Specular bounce on a letter. No color acc.
      direction = direction + normal * (normal % direction * -2);
      origin = sampledPosition + direction * 0.1f;
      attenuation = attenuation * 0.2f; // Attenuation via distance traveled.
    }
    if (hitType == HIT_WALL) { // Wall hit uses color yellow?
      float incidence = normal % lightDirection;
      float p = 6.283185f * randomVal();
      float c = randomVal();
      float s = sqrtf(1 - c);
      float g = normal.z < 0 ? -1 : 1;
      float u = __fdividef(-1, (g + normal.z));
      float v = normal.x * normal.y * u;
      float cosp;
      float sinp;
      __sincosf(p, &sinp, &cosp); 
      direction =
          Vec(v, g + normal.y * normal.y * u, -normal.y) * (cosp * s) +
          Vec(1 + g * normal.x * normal.x * u, g * v, -g * normal.x) *
              (sinp * s) +
          normal * sqrtf(c);
      origin = sampledPosition + direction * 0.1f;
      attenuation = attenuation * 0.2f;
      if (incidence > 0.0f &&
          RayMarching(sampledPosition + normal * 0.1f, lightDirection,
                      sampledPosition, normal) == HIT_SUN)
        color = color + attenuation * Vec(500, 400, 100) * incidence;
    }
    if (hitType == HIT_SUN) { //
      color = color + attenuation * Vec(50, 80, 100);
      break; // Sun Color
    }
  }
  return color;
}


__global__ void GetColor(unsigned char *img, int samplesCount) {
  int x = blockIdx.x;
  int y = threadIdx.x;  

  hiprand_init( blockIdx.x * threadIdx.x, 0, 0 , &states[threadIdx.x]);

  const Vec position(-22.0f, 5.0f, 25.0f);
  const Vec goal = !(Vec(-3.0f, 4.0f, 0.0f) + position * -1.0f);
  const Vec left = !Vec(goal.z, 0, -goal.x) * (1.0f / w);

  // Cross-product to get the up vector
  const Vec up(goal.y *left.z - goal.z * left.y, goal.z *left.x - goal.x * left.z,
       goal.x *left.y - goal.y * left.x);

  Vec color;
  for (int p = samplesCount; p--;) {
    color = color + Trace(position, 
                          !(goal + left * (x - w / 2.0f + randomVal()) + up * (y - h / 2.0f + randomVal())));
  }

  // Reinhard tone mapping
  color = color * (1.0f / samplesCount) + 14.0f / 241.0f;
  Vec o = color + 1.0f;
  color = Vec(color.x / o.x, color.y / o.y, color.z / o.z) * 255.0f;
  img[(y * w + x) * BPP]     = color.x;
  img[(y * w + x) * BPP + 1] = color.y;
  img[(y * w + x) * BPP + 2] = color.z;
  
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char **argv) {
DWORD start_time = GetTickCount();

  unsigned char *dev_bitmap;
  hipMalloc( (void**)&dev_bitmap, (w * h * BPP) );
  gpuErrchk( hipPeekAtLastError() );

  int samplesCount = 1 << 10;
  if (argc > 1 ) {
    samplesCount = atoi(argv[1]);
  }  

  GetColor<<<w,h>>>(dev_bitmap, samplesCount); 
  gpuErrchk( hipDeviceSynchronize() );

  char* bitmap = new char[w * h * BPP];
  hipMemcpy( bitmap, dev_bitmap, (w * h * BPP), hipMemcpyDeviceToHost );
  gpuErrchk( hipPeekAtLastError() );
 
  DWORD elapsed_ms = GetTickCount() - start_time;
  fprintf(stderr, "Time: %dms\n", elapsed_ms);

  printf("P6 %d %d 255 ", w, h);
  char* c = bitmap;
  for (int y = h; y--;) {
    for (int x = w; x--;) {
      c = &bitmap[y * w * BPP  + x * BPP];
      printf("%c%c%c", c[0], c[1], c[2]);
      c += BPP;
    }
  }

  delete bitmap;
  return EXIT_SUCCESS;
}

// Andrew Kensler
